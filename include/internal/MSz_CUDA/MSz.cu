#include "hip/hip_runtime.h"
#include <iostream>
#include <float.h> 
#include <hipblas.h>
#include <fstream>
#include <sstream>
#include "../../api/MSz.h"
#include <vector>
#include <cstdlib>
#include <stdio.h>
#include <unordered_map>
#include <random>
#include <iostream>
#include <cstring> 
#include <chrono> 
#include <hip/hip_runtime.h>
#include <string>
#include <unordered_set>
#include <set>
#include <map>
#include <algorithm>
#include <numeric>
#include <utility>
#include <iomanip>
#include <chrono>
#include <thrust/device_vector.h>
using std::count;
using std::cout;
using std::endl;


__device__ double* decp_data;
__device__ double* decp_data_copy ;
__device__ int directions1[78] = {
    1, 0, 0, -1, 0, 0,   
    0, 1, 0, 0, -1, 0,
    0, 0, 1, 0, 0, -1,
    1, 1, 0, 1, -1, 0,  
    -1, 1, 0, -1, -1, 0,
    1, 0, 1, 1, 0, -1,
    -1, 0, 1, -1, 0, -1,
    0, 1, 1, 0, 1, -1,
    0, -1, 1, 0, -1, -1,
    1, 1, 1, 1, 1, -1,  
    1, -1, 1, 1, -1, -1,
    -1, 1, 1, -1, 1, -1,
    -1, -1, 1, -1, -1, -1
};

__device__ int width;
__device__ int height;
__device__ int depth;
__device__ int num;
__device__ int* adjacency;
__device__ double* d_deltaBuffer1;
__device__ int* number_array;
__device__ int* all_max; 
__device__ int* all_min;
__device__ int* all_p_max; 
__device__ int* all_p_min;
__device__ int* unsigned_n;
__device__ int count_max;
__device__ int count_min;
__device__ int count_f_max;
__device__ int count_f_min;
__device__ int count_p_max;
__device__ int count_p_min;
__device__ int count_false_label;
__device__ int* maxi;

__device__ double bound;
__device__ int edit_count;
__device__ int* or_maxi;
__device__ int* or_mini;
__device__ double* d_deltaBuffer;
__device__ int* id_array;
__device__ int* or_label;
__device__ int* dec_label;

__device__ double* input_data;
__device__ int* de_direction_as;
__device__ int* de_direction_ds;
__device__ int maxNeighbors = 12;

__device__ int direction_to_index_mapping_cuda[26][3] = 
{
    {1, 0, 0}, {-1, 0, 0},   
    {0, 1, 0}, {0, -1, 0},
    {0, 0, 1}, {0, 0, -1},
    {1, 1, 0}, {1, -1, 0},  
    {-1, 1, 0}, {-1, -1, 0},
    {1, 0, 1}, {1, 0, -1},
    {-1, 0, 1}, {-1, 0, -1},
    {0, 1, 1}, {0, 1, -1},
    {0, -1, 1}, {0, -1, -1},
    {1, 1, 1}, {1, 1, -1},  
    {1, -1, 1}, {1, -1, -1},
    {-1, 1, 1}, {-1, 1, -1},
    {-1, -1, 1}, {-1, -1, -1}
};   


__device__ int getDirection(int x, int y, int z){
    
    for (int i = 0; i < maxNeighbors; ++i) {
        if (direction_to_index_mapping_cuda[i][0] == x && direction_to_index_mapping_cuda[i][1] == y && direction_to_index_mapping_cuda[i][2] == z) {
            return i+1;  
        }
    }
    return -1;  


}


__device__ int from_direction_to_index(int cur, int direc){
    
    if (direc==-1) return cur;
    int x = cur % width;
    int y = (cur / width) % height;
    int z = (cur/(width * height))%depth;
    
    if (direc >= 1 && direc <= maxNeighbors) {
        int delta_row = direction_to_index_mapping_cuda[direc-1][0];
        int delta_col = direction_to_index_mapping_cuda[direc-1][1];
        int delta_dep = direction_to_index_mapping_cuda[direc-1][2];
        
        
        int next_row = x + delta_row;
        int next_col = y + delta_col;
        int next_dep = z + delta_dep;
        
        return next_row + next_col * width + next_dep* (height * width);
    }
    else {
        return -1;
    }
    // return 0;
};

__global__ void find_direction (int type=0){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index>=num){
        return;
    }
    
    double *data;
    int *direction_as;
    int *direction_ds;
    if(type==0){
        data = decp_data;
        direction_as = de_direction_as;
        direction_ds = de_direction_ds;
    }
    else{
        data = input_data;
        direction_as = or_maxi;
        direction_ds = or_mini;
    }
    
        
        
    int largetst_index = index;

    
    for(int j =0;j<maxNeighbors;++j){
        int i = adjacency[index*maxNeighbors+j];
        
        if(i==-1){
            continue;
        }
        
        if((data[i]>data[largetst_index] or (data[i]==data[largetst_index] and i>largetst_index))){
            
            largetst_index = i;
            // }
            
        };
    };
    int x_diff = (largetst_index % width) - (index % width);
    int y_diff = (largetst_index / (width)) % height - (index / (width)) % height;
    int z_diff = (largetst_index /(width * height)) % depth - (index /(width * height)) % depth;
    
    direction_as[index] = getDirection(x_diff, y_diff,z_diff);
    largetst_index = index;
    for(int j =0;j<maxNeighbors;++j){
        int i = adjacency[index*maxNeighbors+j];
        
        if(i==-1){
            continue;
        }
        
        if((data[i]<data[largetst_index] or (data[i]==data[largetst_index] and i<largetst_index))){
            
            largetst_index = i;
        };
    };
    
    
    x_diff = (largetst_index % width) - (index % width);
    y_diff = (largetst_index / (width)) % height - (index / (width)) % height;
    z_diff = (largetst_index /(width * height)) % depth - (index /(width * height)) % depth;
    direction_ds[index] = getDirection(x_diff, y_diff,z_diff);
    
    return;

};

__global__ void iscriticle(){
        
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        
        if(i>=num){
            
            return;
        }
        
        bool is_maxima = true;
        bool is_minima = true;
        
        for (int index=0;index<maxNeighbors;index++) {
            int j = adjacency[i*maxNeighbors+index];
            if(j==-1){
                continue;
            }
              
            if (decp_data[j] > decp_data[i]) {
                
                is_maxima = false;
                
                break;
            }
            else if(decp_data[j] == decp_data[i] and j>i){
                is_maxima = false;
                break;
            }
        }
        for (int index=0;index< maxNeighbors;index++) {
            int j = adjacency[i*maxNeighbors+index];
            if(j==-1){
                continue;
            }
            if (decp_data[j] < decp_data[i]) {
                is_minima = false;
                break;
            }
            else if(decp_data[j] == decp_data[i] and j<i){
                is_minima = false;
                break;
            }
        }
        
        
        if((is_maxima && or_maxi[i]!=-1) or (!is_maxima && or_maxi[i]==-1)){
            int idx_fp_max = atomicAdd(&count_f_max, 1);
            
            all_max[idx_fp_max] = i;
            
        }
        
        else if ((is_minima && or_mini[i]!=-1) or (!is_minima && or_mini[i]==-1)) {
            int idx_fp_min = atomicAdd(&count_f_min, 1);
            
            all_min[idx_fp_min] = i;
            
        } 
        
       
        
}

__global__ void get_wrong_index_path(){

    
    int i = threadIdx.x + blockIdx.x * blockDim.x;
        
    if(i>=num){
        
        return;
    }
    
    if (or_label[i * 2 + 1] != dec_label[i * 2 + 1]) {
        int idx_fp_max = atomicAdd(&count_p_max, 1);
        all_p_max[idx_fp_max] = i;
            
    }
    if (or_label[i * 2] != dec_label[i * 2]) {
        int idx_fp_min = atomicAdd(&count_p_min, 1);
        all_p_min[idx_fp_min] = i;
    }
    
    

    return;
};

__global__ void get_wrong_index_count(){

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i>=num){
        
        return;
    }
    
    if (or_label[i * 2 + 1] != dec_label[i * 2 + 1]) {
        atomicAdd(&count_false_label, 1);
    }
    else if (or_label[i * 2] != dec_label[i * 2]) {
        atomicAdd(&count_false_label, 1);
    }
    
    

    return;
};


__global__ void computeAdjacency() {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < num) {
        
        int y = (i / (width)) % height; // Get the x coordinate
        int x = i % width; // Get the y coordinate
        int z = (i / (width * height)) % depth;
        int neighborIdx = 0;
        
        for (int d = 0; d < maxNeighbors; d++) {
            
            int dirX = directions1[d * 3];     
            int dirY = directions1[d * 3 + 1]; 
            int dirZ = directions1[d * 3 + 2]; 
            int newX = x + dirX;
            int newY = y + dirY;
            int newZ = z + dirZ;
            int r = newX + newY * width + newZ* (height * width); // Calculate the index of the adjacent vertex
            
            if (newX >= 0 && newX < width && newY >= 0 && newY < height && r < width*height*depth && newZ<depth && newZ>=0 ) {
                
                adjacency[i * maxNeighbors + neighborIdx] = r;
                neighborIdx++;

            }
        }

        // Fill the remaining slots with -1 or another placeholder value
        
        for (int j = neighborIdx; j < maxNeighbors; ++j) {
            adjacency[i * maxNeighbors + j] = -1;
        }
    }
}



__device__ double atomicCASDouble(double* address, double val) {
   
    uint64_t* address_as_ull = (uint64_t*)address;
    uint64_t old_val_as_ull = *address_as_ull;
    uint64_t new_val_as_ull = __double_as_longlong(val);
    uint64_t assumed;


    assumed = old_val_as_ull;
    
    old_val_as_ull = atomicCAS((unsigned long long int*)address_as_ull, (unsigned long long int)assumed, (unsigned long long int)new_val_as_ull);
    return __longlong_as_double(old_val_as_ull);
}

__device__ int swap(int index, double delta){
    int update_successful = 0;
    
    while (update_successful==0) {
        double current_value = d_deltaBuffer[index];
        if (-delta > current_value) {
            double swapped = atomicCASDouble(&d_deltaBuffer[index], delta);
            if (swapped == current_value) {
                update_successful = 1;
                
            } 
        } else {
            update_successful = 1; 
        }
    }
}

__global__ void fix_maxi_critical(int direction, int cnt){
    int index_f = blockIdx.x * blockDim.x + threadIdx.x;
        
    int index;
    int next_vertex;

    if (direction == 0 && index_f<count_f_max){
        
        index = all_max[index_f];
        // if vertex is a regular point.
        if (or_maxi[index]!=-1){
            
            // find its largest neighbor
            
            next_vertex = from_direction_to_index(index,or_maxi[index]);
            
            double d = ((input_data[index] - bound) + decp_data[index]) / 2.0 - decp_data[index];
            
            if(decp_data[index]<decp_data[next_vertex] or (decp_data[index]==decp_data[next_vertex] and index<next_vertex)){
                return;
            }

            
            double oldValue = d_deltaBuffer[index];
            
            if (d > oldValue) {
                swap(index, d);
            }  

            return;
            
            
            
        
        }
        else{
            // if is a maximum in the original data;
            
            int largest_index = from_direction_to_index(index, de_direction_as[index]);
            
            if(decp_data[index]>decp_data[largest_index] or(decp_data[index]==decp_data[largest_index] and index>largest_index)){
                return;
            }

            double d = ((input_data[largest_index] - bound) + decp_data[largest_index]) / 2.0 - decp_data[largest_index];
            
            double oldValue = d_deltaBuffer[largest_index];
            if (d > oldValue) {
                swap(largest_index, d);
            }  

            return;
        }
        
        
    
    }
    
    else if (direction != 0 && index_f<count_f_min){
        index = all_min[index_f];
        
        if (or_mini[index]!=-1){
           
            
            int next_vertex= from_direction_to_index(index,or_mini[index]);
            

            double d = ((input_data[next_vertex] - bound) + decp_data[index]) / 2.0 - decp_data[next_vertex];
            
            if(decp_data[index]>decp_data[next_vertex] or (decp_data[index]==decp_data[next_vertex] and index>next_vertex)){
                return;
            }

            double oldValue = d_deltaBuffer[next_vertex];
            if (d > oldValue) {
                swap(next_vertex, d);
            }  

            return;
       
        
        }
    
        else{
            
            int largest_index = from_direction_to_index(index,de_direction_ds[index]);
            
            if(decp_data[index]<decp_data[largest_index] or (decp_data[index]==decp_data[largest_index] and index<largest_index)){
                
                return;
            }
            
            double d = ((input_data[index] - bound) + decp_data[index]) / 2.0 - decp_data[index];
            
            double oldValue = d_deltaBuffer[index];
            if (d > oldValue) {
                swap(index, d);
            }  

            return; 
        }

        
    }    
    

    

    return;
}


__global__ void initializeKernel(double value) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid<num){
        d_deltaBuffer[tid] = -4.0 * bound;
    }

}

__global__ void fixpath(int direction){
    int index_f = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(direction == 0){
        if(index_f<count_p_max){

        
        int index = all_p_max[index_f];
        int cur = index;
        while (or_maxi[cur] == de_direction_as[cur]){
            int next_vertex =  from_direction_to_index(cur,de_direction_as[cur]);
            
            if(de_direction_as[cur]==-1 && next_vertex == cur){
                cur = -1;
                break;
            }
            if(next_vertex == cur){
                cur = next_vertex;
                break;
            };
            
            cur = next_vertex;
        }

        int start_vertex = cur;
        
        
        if (start_vertex==-1) return;
        else{
            
            int false_index= from_direction_to_index(cur,de_direction_as[cur]);
            int true_index= from_direction_to_index(cur, or_maxi[cur]);
            if(false_index==true_index) return;

            double d = ((input_data[false_index] - bound) + decp_data[false_index]) / 2.0 - decp_data[false_index];
             
            double oldValue = d_deltaBuffer[false_index];
            if (d > oldValue) {
                swap(false_index, d);
            }  

            return;
        }
        }
    }

    else 
    {
        if(index_f<count_p_min){
            
        int index = all_p_min[index_f];
        int cur = index;
        
        
        while (or_mini[cur] == de_direction_ds[cur]){
            
            int next_vertex = from_direction_to_index(cur,de_direction_ds[cur]);
            
            
            if (next_vertex == cur){
                cur = next_vertex;
                break;
            }
            cur = next_vertex;

            
                
        }
    
        int start_vertex = cur;
        
        if (start_vertex==-1) return;
        
        else{
            
            int false_index= from_direction_to_index(cur,de_direction_ds[cur]);
            int true_index= from_direction_to_index(cur, or_mini[cur]);
            if(false_index==true_index) return;

            double d = ((input_data[true_index] - bound) + decp_data[true_index]) / 2.0 - decp_data[true_index];
            double oldValue = d_deltaBuffer[true_index];
            if (d > oldValue) {
                swap(true_index, d);
            }  

            return;
        }
    }
    }
    return;
};

__global__ void applyDeltaBuffer() {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < num) {
        if(d_deltaBuffer[tid] > -4.0 * bound){
            
            if(abs(d_deltaBuffer[tid]) > 1e-15) decp_data[tid] += d_deltaBuffer[tid];
            else decp_data[tid] = input_data[tid] - bound;
        }

        
    }
    
}


__global__ void getlabel(int *un_sign_ds, int *un_sign_as, int type=0){
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int *direction_as;
    int *direction_ds;
    int *label;
    
    if(i>=num){
        return;
    }
    
    if(type==0){
        direction_as = de_direction_as;
        direction_ds = de_direction_ds;
        label = dec_label;
    }
    else{
        direction_as = or_maxi;
        direction_ds = or_mini;
        label = or_label;
    }
    
    int cur = label[i*2+1];
    
    
        int next_vertex;
        
        if (cur!=-1 and direction_as[cur]!=-1){
            
            int direc = direction_as[cur];
            
            
            next_vertex = from_direction_to_index(cur, direc);
            
            
            if(label[next_vertex*2+1] == -1){
                label[i*2+1] = next_vertex;
                
            }
            
            else{
                
                label[i*2+1] = label[next_vertex*2+1];
                
                
            }
            
            if (direction_as[label[i*2+1]] != -1){
                
                *un_sign_as+=1;  
                
            }
            
        }
    
    
    
    
        cur = label[i*2];
        int next_vertex1;
        
        
        if (cur!=-1 and label[cur*2]!=-1){
            
            int direc = direction_ds[cur];
            
            next_vertex1 = from_direction_to_index(cur, direc);
            
            if(label[next_vertex1*2] == -1){
                label[i*2] = next_vertex1;
                
            }
            
            else if(label[label[next_vertex1*2]*2] == -1){
                label[i*2] = label[next_vertex1*2];  
            }
            
            else if(direction_ds[i]!=-1){
               
                if(label[next_vertex1*2]!=-1){
                    label[i*2] = label[next_vertex1*2];
                }
                
                else{

                    label[i*2] = next_vertex1;
                }
                
                
            }
            
            if (direction_ds[label[i*2]]!=-1){
                *un_sign_ds+=1;
                }
            } 
        
        
    return;

}


__global__ void initializeWithIndex(int size, int type=0) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int* label;
    if (index < size) {
        int *direction_ds;
        int *direction_as;
        if(type==0){
            direction_ds = de_direction_ds;
            direction_as = de_direction_as;
            label = dec_label;
        }
        else{
            direction_ds = or_mini;
            direction_as = or_maxi;
            label = or_label;
        }

        if(direction_ds[index]!=-1){
            label[index*2] = index;
            
        }
        else{
            label[index*2] = -1;
        }

        if(direction_as[index]!=-1){
            label[index*2+1] = index;
        }
        else{
            label[index*2+1] = -1;
        }
    }
}

__global__ void change_mode(int neighbor_number)
{
    if(neighbor_number != 0)
    {
        maxNeighbors = 26;
    }
}


int fix_process(std::vector<int> *a,std::vector<int> *b,
                std::vector<int> *c,std::vector<int> *d,
                std::vector<double> *input_data1,std::vector<double> *decp_data1,
                std::vector<int>* dec_label1,std::vector<int>* or_label1, 
                int width1, int height1, int depth1, 
                double bound1, 
                int preserve_min, int preserve_max, 
                int preserve_path, int neighbor_number, int device_id){
    hipError_t cudaStatus = hipSetDevice(device_id);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_NO_AVAILABLE_GPU;
    }
    int *temp, *temp1, *d_data, *or_l, *dec_l;
    
    double *temp3, *temp4;
    int max_n = 12;
    if(neighbor_number!=0) max_n = 26;
    int num1 = width1*height1*depth1;
    
    int h_un_sign_as = num1;
    int h_un_sign_ds = num1;
  
    
    int *un_sign_as;
    hipMalloc((void**)&un_sign_as, sizeof(int));
    hipMemset(un_sign_as, 0, sizeof(int));

    int *un_sign_ds;
    hipMalloc((void**)&un_sign_ds, sizeof(int));
    hipMemset(un_sign_ds, 0, sizeof(int));

    
    
    std::vector<int> h_all_p_max(num1);
    std::vector<int> h_all_p_min(num1);


    hipMemcpyToSymbol(HIP_SYMBOL(width), &width1, sizeof(int), 0, hipMemcpyHostToDevice);
    
    hipMemcpyToSymbol(HIP_SYMBOL(height), &height1, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(depth), &depth1, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(num), &num1, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(bound), &bound1, sizeof(double), 0, hipMemcpyHostToDevice);
    
    cudaStatus = hipMalloc(&temp, num1 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    cudaStatus = hipMalloc(&temp1, num1 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    cudaStatus = hipMalloc(&temp3, num1  * sizeof(double));
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }

    cudaStatus = hipMalloc(&temp4, num1  * sizeof(double));
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    
    cudaStatus = hipMalloc(&d_data, num1 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    cudaStatus = hipMalloc(&or_l, num1 * 2  * sizeof(int));
    hipMalloc(&dec_l, num1 * 2 * sizeof(int));
    
    
    hipEvent_t start, stop;

    hipEventCreate(&start);

    hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    

    hipMemcpy(temp3, input_data1->data(), num1 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(temp4, decp_data1->data(), num1 * sizeof(double), hipMemcpyHostToDevice);
    
    change_mode<<<1,1>>>(neighbor_number);
    hipDeviceSynchronize();
    

    int *d_temp;  
    size_t size = num1 * sizeof(int);

    cudaStatus = hipMalloc(&d_temp, size);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }

    hipMemcpyToSymbol(HIP_SYMBOL(all_max), &d_temp, sizeof(int*));
    

    
    int *d_temp1;  
    size_t size1 = num1 * sizeof(int);
    cudaStatus = hipMalloc(&d_temp1, size1);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(all_min), &d_temp1, sizeof(int*));

    int *p_temp; 
    cudaStatus = hipMalloc(&p_temp, size1);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(all_p_min), &p_temp, sizeof(int*));

    int *p_temp1;  
    
    cudaStatus = hipMalloc(&p_temp1, size1);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(all_p_max), &p_temp1, sizeof(int*));

    int *d_temp2;  
    size_t size4 = num1  * sizeof(int);
    
    cudaStatus = hipMalloc(&d_temp2, size4);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(de_direction_as), &d_temp2, sizeof(int*));
    hipMemcpyToSymbol(HIP_SYMBOL(or_label), &or_l, sizeof(int*));
    hipMemcpyToSymbol(HIP_SYMBOL(dec_label), &dec_l, sizeof(int*));

    int *d_temp3;  
    size_t size3 = num1 * sizeof(int);
    cudaStatus = hipMalloc(&d_temp3, size3);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    
    hipMemcpyToSymbol(HIP_SYMBOL(de_direction_ds), &d_temp3, sizeof(int*));
    hipMemcpyToSymbol(HIP_SYMBOL(or_maxi), &temp, sizeof(int*));
    hipMemcpyToSymbol(HIP_SYMBOL(or_mini), &temp1, sizeof(int*));
    hipMemcpyToSymbol(HIP_SYMBOL(input_data), &temp3, sizeof(double*));
    hipMemcpyToSymbol(HIP_SYMBOL(decp_data), &temp4, sizeof(double*));

    
    
    dim3 blockSize(256);
    dim3 gridSize((num1 + blockSize.x - 1) / blockSize.x);
    
    int* tempDevicePtr = nullptr;
    size_t arraySize = num1*max_n; 
    cudaStatus = hipMalloc(&tempDevicePtr, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(adjacency), &tempDevicePtr, sizeof(tempDevicePtr));
   

    computeAdjacency<<<gridSize, blockSize>>>();
    hipDeviceSynchronize();
    find_direction<<<gridSize, blockSize>>>(1);
    
    
    
   
    double init_value = -2*bound1;
    double* buffer_temp;
    cudaStatus = hipMalloc(&buffer_temp, num1  * sizeof(double));
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(d_deltaBuffer), &buffer_temp, sizeof(double*));

    double* array_temp;
    cudaStatus = hipMalloc(&array_temp, num1  * sizeof(int));
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(id_array), &array_temp, sizeof(int*));
   
    find_direction<<<gridSize, blockSize>>>();
   
    
    int initialValue = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(count_f_max), &initialValue, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(count_f_min), &initialValue, sizeof(int));
    iscriticle<<<gridSize,blockSize>>>();
    
    
    // double h_s[num1];
    int host_count_f_max;
    hipMemcpyFromSymbol(&host_count_f_max, HIP_SYMBOL(count_f_max), sizeof(int), 0, hipMemcpyDeviceToHost);
    
    int host_count_f_min;
    hipMemcpyFromSymbol(&host_count_f_min, HIP_SYMBOL(count_f_min), sizeof(int), 0, hipMemcpyDeviceToHost);
    
    
    int cnt  = 0;
    
    std::vector<int> h_all_max(num1);

    if(preserve_max == 0) host_count_f_max = 0;
    if(preserve_min == 0) host_count_f_min = 0;
    
    
    while(host_count_f_min>0 || host_count_f_max>0){
            
            
            initializeKernel<<<gridSize, blockSize>>>(init_value);
            
            hipDeviceSynchronize();
            
            dim3 blockSize1(256);
            dim3 gridSize1((host_count_f_max + blockSize1.x - 1) / blockSize1.x);
            
            if(preserve_max == 1)
            {
                fix_maxi_critical<<<gridSize1, blockSize1>>>(0,cnt);
            }   
 
            dim3 blocknum(256);
            dim3 gridnum((host_count_f_min + blocknum.x - 1) / blocknum.x);
            if(preserve_min == 1)
            {
                fix_maxi_critical<<<gridnum, blocknum>>>(1,cnt);
            }
            
            applyDeltaBuffer<<<gridSize, blockSize>>>();
            hipDeviceSynchronize();
            hipMemcpyToSymbol(HIP_SYMBOL(count_f_max), &initialValue, sizeof(int));
            hipMemcpyToSymbol(HIP_SYMBOL(count_f_min), &initialValue, sizeof(int));
            
            hipDeviceSynchronize();
            
            iscriticle<<<gridSize, blockSize>>>();
            find_direction<<<gridSize,blockSize>>>();

            hipMemcpyFromSymbol(&host_count_f_max, HIP_SYMBOL(count_f_max), sizeof(int), 0, hipMemcpyDeviceToHost);
            hipMemcpyFromSymbol(&host_count_f_min, HIP_SYMBOL(count_f_min), sizeof(int), 0, hipMemcpyDeviceToHost);
            
            hipDeviceSynchronize();
            if(preserve_max == 0) host_count_f_max = 0;
            if(preserve_min == 0) host_count_f_min = 0;
            
    }

    
    if(preserve_path ==0 || preserve_max == 0 || preserve_min == 0) 
    {
        hipMemcpy(decp_data1->data(), temp4, num1 * sizeof(double), hipMemcpyDeviceToHost);
        return MSZ_ERR_NO_ERROR;
    }
    

    initializeWithIndex<<<gridSize, blockSize>>>(num1,0);
    initializeWithIndex<<<gridSize, blockSize>>>(num1,1);
    
   
    while(h_un_sign_as>0 or h_un_sign_ds>0){
        
        int zero = 0;
        int zero1 = 0;

        
        hipMemcpy(un_sign_as, &zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(un_sign_ds, &zero1, sizeof(int), hipMemcpyHostToDevice);
        getlabel<<<gridSize, blockSize>>>(un_sign_as,un_sign_ds,0);
        
        hipMemcpy(&h_un_sign_as, un_sign_as, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&h_un_sign_ds, un_sign_ds, sizeof(int), hipMemcpyDeviceToHost);
        
        
    }   
    
    
    
    
    
    h_un_sign_as = num1;
    h_un_sign_ds = num1;
    while(h_un_sign_as>0 or h_un_sign_ds>0){
        
        int zero = 0;
        int zero1 = 0;


        hipMemcpy(un_sign_as, &zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(un_sign_ds, &zero1, sizeof(int), hipMemcpyHostToDevice);
        getlabel<<<gridSize, blockSize>>>(un_sign_as,un_sign_ds,1);
        
        hipMemcpy(&h_un_sign_as, un_sign_as, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&h_un_sign_ds, un_sign_ds, sizeof(int), hipMemcpyDeviceToHost);
        
        
        
    }
    
    
    hipMemcpy(dec_label1->data(), dec_l, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(or_label1->data(), or_l, num1 * sizeof(int), hipMemcpyDeviceToHost);
    
    
    
    hipMemcpyToSymbol(HIP_SYMBOL(count_p_max), &initialValue, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(count_p_min), &initialValue, sizeof(int));
    get_wrong_index_path<<<gridSize, blockSize>>>();

    int host_count_p_max;
    
    hipMemcpyFromSymbol(&host_count_p_max, HIP_SYMBOL(count_p_max), sizeof(int), 0, hipMemcpyDeviceToHost);
    
    int host_count_p_min;
    hipMemcpyFromSymbol(&host_count_p_min, HIP_SYMBOL(count_p_min), sizeof(int), 0, hipMemcpyDeviceToHost);
    
    
    while(host_count_p_min>0 or host_count_p_max>0 or host_count_f_min>0 or host_count_f_max>0){
        
     

        initializeKernel<<<gridSize, blockSize>>>(init_value);
        dim3 blockSize2(256);
        dim3 gridSize2((host_count_p_max + blockSize2.x - 1) / blockSize2.x);


        
        fixpath<<<gridSize2, blockSize2>>>(0);
        hipDeviceSynchronize();
        
        
        
        dim3 blockSize3(256);
        dim3 gridSize3((host_count_p_min + blockSize3.x - 1) / blockSize3.x);
        fixpath<<<gridSize3, blockSize3>>>(1);
        hipDeviceSynchronize();

        applyDeltaBuffer<<<gridSize, blockSize>>>();
        hipDeviceSynchronize();
        
        find_direction<<<gridSize, blockSize>>>();
        hipDeviceSynchronize();

        

        hipMemcpyToSymbol(HIP_SYMBOL(count_f_max), &initialValue, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(count_f_min), &initialValue, sizeof(int));
        
        iscriticle<<<gridSize, blockSize>>>();
        hipDeviceSynchronize();

        hipMemcpyFromSymbol(&host_count_f_max, HIP_SYMBOL(count_f_max), sizeof(int), 0, hipMemcpyDeviceToHost);
        hipMemcpyFromSymbol(&host_count_f_min, HIP_SYMBOL(count_f_min), sizeof(int), 0, hipMemcpyDeviceToHost);
        
        while(host_count_f_max>0 or host_count_f_min>0){
            dim3 blockSize1(256);
            dim3 gridSize1((host_count_f_max + blockSize1.x - 1) / blockSize1.x);
            
            initializeKernel<<<gridSize, blockSize>>>(init_value);
            
            fix_maxi_critical<<<gridSize1, blockSize1>>>(0,cnt);
            
            hipDeviceSynchronize();
            // hipDeviceSynchronize();
            
            
            dim3 blocknum(256);
            dim3 gridnum((host_count_f_min + blocknum.x - 1) / blocknum.x);
            
            fix_maxi_critical<<<gridnum, blocknum>>>(1,cnt);
            hipDeviceSynchronize();
            
            
            
            hipMemcpyToSymbol(HIP_SYMBOL(count_f_max), &initialValue, sizeof(int));
            hipMemcpyToSymbol(HIP_SYMBOL(count_f_min), &initialValue, sizeof(int));
            
            applyDeltaBuffer<<<gridSize, blockSize>>>();
            find_direction<<<gridSize,blockSize>>>();
            iscriticle<<<gridSize, blockSize>>>();
            
            hipMemcpyFromSymbol(&host_count_f_max, HIP_SYMBOL(count_f_max), sizeof(int), 0, hipMemcpyDeviceToHost);
            
            hipMemcpyFromSymbol(&host_count_f_min, HIP_SYMBOL(count_f_min), sizeof(int), 0, hipMemcpyDeviceToHost);
            
            hipDeviceSynchronize();
            
           
        }
        
        initializeWithIndex<<<gridSize, blockSize>>>(num1,0);
        
        h_un_sign_as = num1;
        h_un_sign_ds = num1;
        
        while(h_un_sign_as>0 or h_un_sign_ds>0){
        
            int zero = 0;
            int zero1 = 0;

            
            hipMemcpy(un_sign_as, &zero, sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(un_sign_ds, &zero1, sizeof(int), hipMemcpyHostToDevice);
            getlabel<<<gridSize, blockSize>>>(un_sign_as,un_sign_ds,0);
            
            hipMemcpy(&h_un_sign_as, un_sign_as, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&h_un_sign_ds, un_sign_ds, sizeof(int), hipMemcpyDeviceToHost);
           
            
            
        } 
        
        
        hipMemcpyToSymbol(HIP_SYMBOL(count_p_max), &initialValue, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(count_p_min), &initialValue, sizeof(int));

        
        get_wrong_index_path<<<gridSize, blockSize>>>();
       

        hipMemcpyFromSymbol(&host_count_p_max, HIP_SYMBOL(count_p_max), sizeof(int), 0, hipMemcpyDeviceToHost);
        
        hipMemcpyFromSymbol(&host_count_p_min, HIP_SYMBOL(count_p_min), sizeof(int), 0, hipMemcpyDeviceToHost);
        
        hipMemcpyToSymbol(HIP_SYMBOL(count_f_max), &initialValue, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(count_f_min), &initialValue, sizeof(int));

        
        iscriticle<<<gridSize, blockSize>>>();
        


        hipMemcpyFromSymbol(&host_count_f_max, HIP_SYMBOL(count_f_max), sizeof(int), 0, hipMemcpyDeviceToHost);
       
        hipMemcpyFromSymbol(&host_count_f_min, HIP_SYMBOL(count_f_min), sizeof(int), 0, hipMemcpyDeviceToHost);
        
    
    }
    
    
    
    
    hipMemcpy(a->data(), temp, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(b->data(), temp1, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(c->data(), d_temp2, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(d->data(), d_temp3, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(decp_data1->data(), temp4, num1 * sizeof(double), hipMemcpyDeviceToHost);
    

    return MSZ_ERR_NO_ERROR;
}

int count_false_cases(std::vector<int> *a,std::vector<int> *b,
                std::vector<int> *c,std::vector<int> *d,
                std::vector<double> *input_data1,std::vector<double> *decp_data1,
                std::vector<int>* dec_label1,std::vector<int>* or_label1, 
                int width1, int height1, int depth1, int neighbor_number,
                int &wrong_min, int &wrong_max,  int &wrong_labels, int device_id)
{
    hipError_t cudaStatus = hipSetDevice(device_id);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_NO_AVAILABLE_GPU;
    }
    int *temp, *temp1, *d_data, *or_l, *dec_l;
    double *temp3, *temp4;
    int max_n = 12;
    if(neighbor_number!=0) max_n = 26;
    int num1 = width1*height1*depth1;
    
    int h_un_sign_as = num1;
    int h_un_sign_ds = num1;

    
    int *un_sign_as;
    hipMalloc((void**)&un_sign_as, sizeof(int));
    hipMemset(un_sign_as, 0, sizeof(int));

    int *un_sign_ds;
    hipMalloc((void**)&un_sign_ds, sizeof(int));
    hipMemset(un_sign_ds, 0, sizeof(int));


    hipMemcpyToSymbol(HIP_SYMBOL(width), &width1, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(height), &height1, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(depth), &depth1, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(num), &num1, sizeof(int), 0, hipMemcpyHostToDevice);
    
    cudaStatus = hipMalloc(&temp, num1 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    cudaStatus = hipMalloc(&temp1, num1 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    cudaStatus = cudaStatus = hipMalloc(&temp3, num1  * sizeof(double));
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    cudaStatus = hipMalloc(&temp4, num1  * sizeof(double));
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    
    cudaStatus = hipMalloc(&d_data, num1 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    cudaStatus = hipMalloc(&or_l, num1 * 2  * sizeof(int));
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    cudaStatus = hipMalloc(&dec_l, num1 * 2 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    

    

    cudaStatus = hipMemcpy(temp3, input_data1->data(), num1 * sizeof(double), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(temp4, decp_data1->data(), num1 * sizeof(double), hipMemcpyHostToDevice);
    
    
    hipDeviceSynchronize();
    change_mode<<<1,1>>>(neighbor_number);
    hipDeviceSynchronize();
    

    
    int *d_temp;  
    size_t size = num1 * sizeof(int);

    
    cudaStatus = hipMalloc(&d_temp, size);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }

    hipMemcpyToSymbol(HIP_SYMBOL(all_max), &d_temp, sizeof(int*));
    

    
    int *d_temp1;  
    size_t size1 = num1 * sizeof(int);

    
    cudaStatus = hipMalloc(&d_temp1, size1);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(all_min), &d_temp1, sizeof(int*));

    int *p_temp; 
    cudaStatus = hipMalloc(&p_temp, size1);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(all_p_min), &p_temp, sizeof(int*));

    int *p_temp1;  
    cudaStatus = hipMalloc(&p_temp1, size1);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(all_p_max), &p_temp1, sizeof(int*));

    int *d_temp2;  
    size_t size4 = num1  * sizeof(int);
    
    cudaStatus = hipMalloc(&d_temp2, size4);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(de_direction_as), &d_temp2, sizeof(int*));
    hipMemcpyToSymbol(HIP_SYMBOL(or_label), &or_l, sizeof(int*));
    hipMemcpyToSymbol(HIP_SYMBOL(dec_label), &dec_l, sizeof(int*));

    int *d_temp3;  
    size_t size3 = num1 * sizeof(int);

    
    cudaStatus = hipMalloc(&d_temp3, size3);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    
    hipMemcpyToSymbol(HIP_SYMBOL(de_direction_ds), &d_temp3, sizeof(int*));
    hipMemcpyToSymbol(HIP_SYMBOL(or_maxi), &temp, sizeof(int*));
    hipMemcpyToSymbol(HIP_SYMBOL(or_mini), &temp1, sizeof(int*));
    hipMemcpyToSymbol(HIP_SYMBOL(input_data), &temp3, sizeof(double*));
    hipMemcpyToSymbol(HIP_SYMBOL(decp_data), &temp4, sizeof(double*));

    
    
    dim3 blockSize(256);
    dim3 gridSize((num1 + blockSize.x - 1) / blockSize.x);
    int* tempDevicePtr = nullptr;
    size_t arraySize = num1*max_n; 
    cudaStatus = hipMalloc(&tempDevicePtr, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(adjacency), &tempDevicePtr, sizeof(tempDevicePtr));

    computeAdjacency<<<gridSize, blockSize>>>();
    hipDeviceSynchronize();

    find_direction<<<gridSize, blockSize>>>(1);
    
    double* buffer_temp;
    cudaStatus = hipMalloc(&buffer_temp, num1  * sizeof(double));
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(d_deltaBuffer), &buffer_temp, sizeof(double*));

    double* array_temp;
    cudaStatus = hipMalloc(&array_temp, num1  * sizeof(int));
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return MSZ_ERR_OUT_OF_MEMORY;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(id_array), &array_temp, sizeof(int*));
   
    find_direction<<<gridSize, blockSize>>>();
    
   
    
    int initialValue = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(count_f_max), &initialValue, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(count_f_min), &initialValue, sizeof(int));
    iscriticle<<<gridSize,blockSize>>>();
    
    
    
    hipMemcpyFromSymbol(&wrong_max, HIP_SYMBOL(count_f_max), sizeof(int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&wrong_min, HIP_SYMBOL(count_f_min), sizeof(int), 0, hipMemcpyDeviceToHost);
    

    initializeWithIndex<<<gridSize, blockSize>>>(num1,0);
    initializeWithIndex<<<gridSize, blockSize>>>(num1,1);
   
    while(h_un_sign_as>0 or h_un_sign_ds>0){
        
        int zero = 0;
        int zero1 = 0;

        hipMemcpy(un_sign_as, &zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(un_sign_ds, &zero1, sizeof(int), hipMemcpyHostToDevice);
        getlabel<<<gridSize, blockSize>>>(un_sign_as,un_sign_ds,0);
        
        hipMemcpy(&h_un_sign_as, un_sign_as, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&h_un_sign_ds, un_sign_ds, sizeof(int), hipMemcpyDeviceToHost);
        
        
    }   
    
    h_un_sign_as = num1;
    h_un_sign_ds = num1;
    while(h_un_sign_as>0 or h_un_sign_ds>0){
        
        int zero = 0;
        int zero1 = 0;


        hipMemcpy(un_sign_as, &zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(un_sign_ds, &zero1, sizeof(int), hipMemcpyHostToDevice);
        getlabel<<<gridSize, blockSize>>>(un_sign_as,un_sign_ds,1);
        
        hipMemcpy(&h_un_sign_as, un_sign_as, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&h_un_sign_ds, un_sign_ds, sizeof(int), hipMemcpyDeviceToHost);
      
    }

    hipMemcpy(dec_label1->data(), dec_l, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(or_label1->data(), or_l, num1 * sizeof(int), hipMemcpyDeviceToHost);
    
    hipMemcpyToSymbol(HIP_SYMBOL(count_p_max), &initialValue, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(count_p_min), &initialValue, sizeof(int));
    get_wrong_index_count<<<gridSize, blockSize>>>();

    
    hipMemcpyFromSymbol(&wrong_labels, HIP_SYMBOL(count_false_label), sizeof(int), 0, hipMemcpyDeviceToHost);
    
    
    return MSZ_ERR_NO_ERROR;
}
